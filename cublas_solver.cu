#include "hip/hip_runtime.h"
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include "cublas_utils.h"



// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda{m},ldb{k},ldc{n};
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	CUBLAS_CHECK(hipblasCreate(&handle));

	// Do the actual multiplication
	CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));

	// Destroy the handle
	CUBLAS_CHECK(hipblasDestroy(handle));
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
	// Allocate 3 arrays on CPU
	int nr_rows_A{3}, nr_cols_A{3}, nr_rows_B{3}, nr_cols_B{3}, nr_rows_C{3}, nr_cols_C{3};

	// for simplicity we are going to use square arrays
	//nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
	
	float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
	float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

	// Allocate 3 arrays on GPU
	float  *d_A, *d_B, *d_C;
	CUDA_CHECK(hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_B, nr_rows_B * nr_cols_B * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_C, nr_rows_C * nr_cols_C * sizeof(float)));

	// If you already have useful values in A and B you can copy them in GPU:
	hipMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
	GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

	CUDA_CHECK(hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost));
	std::cout << "A =" << std::endl;
	print_matrix(h_A, nr_rows_A, nr_cols_A);
	std::cout << "B =" << std::endl;
	print_matrix(h_B, nr_rows_B, nr_cols_B);

	// Multiply A and B on GPU
	gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

	// Copy (and print) the result on host memory
	CUDA_CHECK(hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost));
	std::cout << "C =" << std::endl;
	print_matrix(h_C, nr_rows_C, nr_cols_C);

	//Free GPU memory
	CUDA_CHECK(hipFree(d_A));
	CUDA_CHECK(hipFree(d_B));
	CUDA_CHECK(hipFree(d_C));	

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
